#include "hip/hip_runtime.h"

#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

__global__ void vectorAdd(int *a, int *b, int *c) {
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}

int main() {
  int a[] = {1, 2, 3, 4, 5, 6, 1, 2, 3, 4, 5, 6, 1, 2, 3, 4, 5, 6, 1, 2, 3,
             4, 5, 6, 1, 2, 3, 4, 5, 6, 1, 2, 3, 4, 5, 6, 1, 2, 3, 4, 5, 6,
             1, 2, 3, 4, 5, 6, 1, 2, 3, 4, 5, 6, 1, 2, 3, 4, 5, 6, 1, 2, 3,
             4, 5, 6, 1, 2, 3, 4, 5, 6, 1, 2, 3, 4, 5, 6, 1, 2, 3, 4, 5, 6};
  int b[] = {4, 5, 6, 1, 2, 3, 4, 5, 6, 1, 2, 3, 4, 5, 6, 1, 2, 3, 4, 5, 6,
             1, 2, 3, 4, 5, 6, 1, 2, 3, 4, 5, 6, 1, 2, 3, 4, 5, 6, 1, 2, 3,
             4, 5, 6, 1, 2, 3, 4, 5, 6, 1, 2, 3, 4, 5, 6, 1, 2, 3, 4, 5, 6,
             1, 2, 3, 4, 5, 6, 1, 2, 3, 4, 5, 6, 1, 2, 3, 4, 5, 6, 1, 2, 3};
  int c[sizeof(a) / sizeof(int)] = {0};

  int *cudaA = 0;
  int *cudaB = 0;
  int *cudaC = 0;

  hipMalloc(&cudaA, sizeof(a));
  hipMalloc(&cudaB, sizeof(b));
  hipMalloc(&cudaC, sizeof(c));

  hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
  hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);

  vectorAdd<<<1, sizeof(a) / sizeof(int)>>>(cudaA, cudaB, cudaC);

  hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

  hipFree(cudaA);
  hipFree(cudaB);
  hipFree(cudaC);

  return 0;
}
